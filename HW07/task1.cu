#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include "matmul.cuh"
#include <random>

int main(int argc, char **argv) {
    if (argc != 3) {
        return -1;
    }

    unsigned int n = std::atoi(argv[1]);
    unsigned int block_dim = std::atoi(argv[2]);

    int *A = new int[n * n];
    int *B = new int[n * n];
    int *C = new int[n * n]; // check if zero init

    // rand
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<int> dis(-10, 10);

    for (unsigned int i = 0; i < n * n; ++i)
    {
        A[i] = dis(gen);
        B[i] = dis(gen);
    }

    // memalloc/set
    int *dA, *dB, *dC;
    hipMalloc(&dA, n * n * sizeof(int));
    hipMalloc(&dB, n * n * sizeof(int));
    hipMalloc(&dC, n * n * sizeof(int));

    hipMemcpy(dA, A, n * n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dB, B, n * n * sizeof(int), hipMemcpyHostToDevice);
    // hipMemcpy(dC, C, n*n * sizeof(float), hipMemcpyHostToDevice);

    // timing
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // kernel call rec
    hipEventRecord(start);
    matmul_1(dA, dB, dC, n, block_dim);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float duration_sec = 0;
    hipEventElapsedTime(&duration_sec, start, stop);
    hipMemcpy(C, dC, n * n * sizeof(int), hipMemcpyDeviceToHost);

    printf("%i\n", C[0]);
    printf("%i\n", C[n * n - 1]);
    printf("%f\n", duration_sec);

    delete[] A;
    delete[] B;
    delete[] C;
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    hipEventDestroy(start);
    hipEventDestroy(stop);




    float *A2 = new float[n * n];
    float *B2 = new float[n * n];
    float *C2 = new float[n * n]; // check if zero init

    // rand
    std::random_device rd2;
    std::mt19937 gen2(rd2());
    std::uniform_real_distribution<float> dis2(-10.0, 10.0);

    for (unsigned int i = 0; i < n * n; ++i)
    {
        A2[i] = dis2(gen2);
        B2[i] = dis2(gen2);
    }

    // memalloc/set
    float *dA2, *dB2, *dC2;
    hipMalloc(&dA2, n * n * sizeof(float));
    hipMalloc(&dB2, n * n * sizeof(float));
    hipMalloc(&dC2, n * n * sizeof(float));

    hipMemcpy(dA2, A2, n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dB2, B2, n * n * sizeof(float), hipMemcpyHostToDevice);
    // hipMemcpy(dC, C, n*n * sizeof(float), hipMemcpyHostToDevice);

    // timing
    hipEvent_t start2;
    hipEvent_t stop2;
    hipEventCreate(&start2);
    hipEventCreate(&stop2);

    // kernel call rec
    hipEventRecord(start2);
    matmul_2(dA2, dB2, dC2, n, block_dim);
    hipEventRecord(stop2);
    hipEventSynchronize(stop2);

    duration_sec = 0;
    hipEventElapsedTime(&duration_sec, start2, stop2);
    hipMemcpy(C2, dC2, n * n * sizeof(float), hipMemcpyDeviceToHost);

    printf("%f\n", C2[0]);
    printf("%f\n", C2[n * n - 1]);
    printf("%f\n", duration_sec);

    delete[] A2;
    delete[] B2;
    delete[] C2;
    hipFree(dA2);
    hipFree(dB2);
    hipFree(dC2);
    hipEventDestroy(start2);
    hipEventDestroy(stop2);


    double *A3 = new double[n * n];
    double *B3 = new double[n * n];
    double *C3 = new double[n * n]; // check if zero init

    // rand
    std::random_device rd3;
    std::mt19937 gen3(rd3());
    std::uniform_real_distribution<double> dis3(-10.0, 10.0);

    for (unsigned int i = 0; i < n * n; ++i)
    {
        A3[i] = dis3(gen3);
        B3[i] = dis3(gen3);
    }

    // memalloc/set
    double *dA3, *dB3, *dC3;
    hipMalloc(&dA3, n * n * sizeof(double));
    hipMalloc(&dB3, n * n * sizeof(double));
    hipMalloc(&dC3, n * n * sizeof(double));

    hipMemcpy(dA3, A3, n * n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dB3, B3, n * n * sizeof(double), hipMemcpyHostToDevice);
    // hipMemcpy(dC, C, n*n * sizeof(float), hipMemcpyHostToDevice);

    // timing
    hipEvent_t start3;
    hipEvent_t stop3;
    hipEventCreate(&start3);
    hipEventCreate(&stop3);

    // kernel call rec
    hipEventRecord(start3);
    matmul_3(dA3, dB3, dC3, n, block_dim);
    hipEventRecord(stop3);
    hipEventSynchronize(stop3);

    duration_sec = 0;
    hipEventElapsedTime(&duration_sec, start3, stop3);
    hipMemcpy(C3, dC3, n * n * sizeof(double), hipMemcpyDeviceToHost);

    printf("%f\n", C3[0]);
    printf("%f\n", C3[n * n - 1]);
    printf("%f\n", duration_sec);

    delete[] A3;
    delete[] B3;
    delete[] C3;
    hipFree(dA3);
    hipFree(dB3);
    hipFree(dC3);
    hipEventDestroy(start3);
    hipEventDestroy(stop3);

    return 0;
}
