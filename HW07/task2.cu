#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include "reduce.cuh"
#include <chrono>
#include <random>


int main(int argc, char **argv) {
    if (argc != 3) {
        return -1;
    }

    unsigned int N = std::atoi(argv[1]);
    unsigned int threads_per_block = std::atoi(argv[2]); 

    float *h_input = new float[N];\
    // rand
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dis(-1.0, 1.0);

    for (unsigned int i = 0; i < N; i++)
    {
        h_input[i] = dis(gen);
    }

    float *d_input, *d_output;
    hipMalloc(&d_input, N * sizeof(float));
    hipMemcpy(d_input, h_input, N * sizeof(float), hipMemcpyHostToDevice);

    unsigned int blocks = (N + threads_per_block * 2 - 1) / (threads_per_block * 2);
    hipMalloc(&d_output, blocks * sizeof(float));

    float *input = d_input;
    float *output = d_output;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    reduce(&input, &output, N, threads_per_block);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    float result;
    hipMemcpy(&result, input, sizeof(float), hipMemcpyDeviceToHost);

    std::cout << result << "\n";
    std::cout << milliseconds << "\n";

    delete[] h_input;
    hipFree(d_input);
    hipFree(d_output);

    return 0;
}
