#include <cstdio>
#include <cstdlib>
#include <random>
#include <hip/hip_runtime.h>
#include "stencil.cuh"

int main(int argc, char **argv)
{
    if (argc != 4)
    {
        return 1;
    }

    int n = std::atoi(argv[1]);
    int R = std::atoi(argv[2]);
    int threads_per_block = std::atoi(argv[3]);

    // lengths
    float *image = new float[n];
    float *mask = new float[2 * R + 1];
    float *output = new float[n];

    // rand
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dis(-1.0, 1.0);

    for (int i = 0; i < n; i++)
    {
        image[i] = dis(gen);
    }
    for (int i = 0; i < 2 * R + 1; i++)
    {
        mask[i] = dis(gen);
    }

    // memalloc/set
    float *dImage, *dMask, *dOutput;
    hipMalloc(&dImage, n * sizeof(float));
    hipMalloc(&dMask, (2 * R + 1) * sizeof(float));
    hipMalloc(&dOutput, n * sizeof(float));

    hipMemcpy(dImage, image, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dMask, mask, (2 * R + 1) * sizeof(float), hipMemcpyHostToDevice);

    // timing
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // kernel call rec
    hipEventRecord(start);
    stencil(dImage, dMask, dOutput, n, R, threads_per_block);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float duration_sec = 0;
    hipEventElapsedTime(&duration_sec, start, stop);
    hipMemcpy(output, dOutput, n * sizeof(float), hipMemcpyDeviceToHost);

    printf("%f\n", output[n - 1]);
    printf("%f\n", duration_sec);

    delete[] image;
    delete[] output;
    delete[] mask;
    hipFree(dImage);
    hipFree(dMask);
    hipFree(dOutput);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}