#include <cstdio>
#include <cstdlib>
#include <random>
#include <hip/hip_runtime.h>
#include "matmul.cuh"

int main(int argc, char **argv)
{

    if (argc != 3)
    {
        return 1;
    }

    int n = std::atoi(argv[1]);
    int threads_per_block = std::atoi(argv[2]);

    float *A = new float[n * n];
    float *B = new float[n * n];
    float *C = new float[n * n]; // check if zero init

    // rand
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dis(-1.0, 1.0);

    for (int i = 0; i < n * n; ++i)
    {
        A[i] = dis(gen);
        B[i] = dis(gen);
    }

    // memalloc/set
    float *dA, *dB, *dC;
    hipMalloc(&dA, n * n * sizeof(float));
    hipMalloc(&dB, n * n * sizeof(float));
    hipMalloc(&dC, n * n * sizeof(float));

    hipMemcpy(dA, A, n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dB, B, n * n * sizeof(float), hipMemcpyHostToDevice);
    // hipMemcpy(dC, C, n*n * sizeof(float), hipMemcpyHostToDevice);

    // timing
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // kernel call rec
    hipEventRecord(start);
    matmul(dA, dB, dC, n * n, threads_per_block); // const float* A, const float* B, float* C, size_t n, unsigned int threads_per_block
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float duration_sec = 0;
    hipEventElapsedTime(&duration_sec, start, stop);
    hipMemcpy(C, dC, n * n * sizeof(float), hipMemcpyDeviceToHost);

    printf("%f\n", C[n * n - 1]);
    printf("%f\n", duration_sec);

    delete[] A;
    delete[] B;
    delete[] C;
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
