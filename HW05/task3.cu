#include <cstdio>
#include <cstdlib>
#include <random>
#include <hip/hip_runtime.h>
#include "vscale.cuh"

int main(int argc, char **argv)
{
    if (argc != 2)
    {
        return 1;
    }

    int n = std::atoi(argv[1]);

    float *hA = new float[n];
    float *hB = new float[n];

    //rand
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> disA(-10.0, 10.0);
    std::uniform_real_distribution<float> disB(0.0, 1.0);

    for (int i = 0; i < n; ++i)
    {
        hA[i] = disA(gen);
        hB[i] = disB(gen);
    }

    //memalloc/set
    float *dA, *dB;
    hipMalloc(&dA, n * sizeof(float));
    hipMalloc(&dB, n * sizeof(float));
    hipMemcpy(dA, hA, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dB, hB, n * sizeof(float), hipMemcpyHostToDevice);

    //timing
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int threadNumInBlock = 512;
    int blocksPerGrid = (n + threadNumInBlock - 1) / threadNumInBlock;

    //kernel call rec
    hipEventRecord(start);
    vscale<<<blocksPerGrid, threadNumInBlock>>>(dA, dB, n);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float duration_sec = 0;
    hipEventElapsedTime(&duration_sec, start, stop);
    hipMemcpy(hB, dB, n * sizeof(float), hipMemcpyDeviceToHost);

    printf("%f\n", duration_sec);
    printf("%f\n", hB[0]);
    printf("%f\n", hB[n - 1]);

    delete[] hA;
    delete[] hB;
    hipFree(dA);
    hipFree(dB);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
